#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "../My_CUDA.hpp"
#include <iostream>



#define SIZE 10


#define check(call) \
{\
 const hipError_t error = call;\
 if (error != hipSuccess) {\
    printf("Error occurs' location is %s : %d\n", __FILE__, __LINE__); \
    printf("Error occurs, Reason is : %s , and Codo is : %d\n", hipGetErrorString(error) , error);\
    exit(1); \
 }\
}

// bool check(hipError_t status) {
//     if (status != hipSuccess) {
//         printf("Error occurs, Reason is : %s , and Codo is : %d", hipGetErrorString(status) , status);
//         return true;
//     }
//     return false;
// }


__global__ void helloworld() {
    printf("Hello World, CUDA!\n");
}


__global__ void add(int* in1, int* in2, int* out) {
    int index = threadIdx.x;
    out[index] = in1[index] + in2[index];
}

int main(void) {
    
    hipError_t status;
    int* in1_h = static_cast<int*>(malloc(SIZE * sizeof(int)));
    int* in2_h = static_cast<int*>(malloc(SIZE * sizeof(int)));
    for (int i = 0; i < SIZE; ++i) {
        in1_h[i] = i;
        in2_h[i] = i;
    }

    int* in1_d = 0;
    int* in2_d = 0;
    status = hipMalloc((void**)&in1_d, SIZE * sizeof(int));
    CUDACHECK(status);
    status = hipMalloc((void**)&in2_d, SIZE * sizeof(int));
    check(status);
    status = hipMemcpy(in1_d, in1_h, SIZE * sizeof(int), hipMemcpyHostToDevice);
    check(status);
    status = hipMemcpy(in2_d, in2_h, SIZE * sizeof(int), hipMemcpyHostToDevice);
    check(status);

    int* out_d = 0;
    status = hipMalloc((void**)&out_d, SIZE * sizeof(int));
    check(status);
     

    dim3 grid(1);
    dim3 block(SIZE);
    add <<< grid, block >>>(in1_d, in2_d, out_d);
    
    status = hipDeviceSynchronize();
    check(status);
    
    int* out_h = static_cast<int*>(malloc(SIZE*sizeof(int)));
    hipMemcpy(out_h, out_d, SIZE*sizeof(int), hipMemcpyDeviceToHost);

    // output
    printf("in1 : \n");
    for(int i = 0; i < SIZE; ++i) {
        printf("%d  ", in1_h[i]);
    }
    printf("\n");

    printf("in2 : \n");
    for(int i = 0; i < SIZE; ++i) {
        printf("%d  ", in2_h[i]);
    }
    printf("\n");

    printf("out : \n");
    for(int i = 0; i < SIZE; ++i) {
        printf("%d  ", out_h[i]);
    }
    printf("\n");


    helloworld<<<1, 10>>>();

Error:
    hipFree(in1_d);
    hipFree(in2_d);
    hipFree(out_d);
    hipDeviceReset();




}
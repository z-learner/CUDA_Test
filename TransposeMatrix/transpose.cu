#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <ctime>
#include <stdlib.h>
#include <Eigen/Dense>

using namespace Eigen;
using namespace std;

#define TILE_DIM 32
#define BLOCK_DIM 32
static bool getError(hipError_t cudaStatus, char* message);

hipError_t transposeWithCuda(float* odata, float* idata, int width, int height);
hipError_t transposeSharedMemWithCuda(float* odata, float* idata, int width, int height);
hipError_t transposeSharedMemNoBankingWithCuda(float* odata, float* idata, int width, int height);




__global__ void transposeNaive(float* odata, float* idata, int width, int height) {
    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

    int in_index = xIndex + width * yIndex;
    int out_index = yIndex + height * xIndex;

    odata[out_index] = idata[in_index];
}

__global__ void transposeSharedMem(float* odata, float* idata, int width, int height) {
    __shared__ float shMat[TILE_DIM][TILE_DIM];
    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    int in_index = xIndex + width * yIndex;
    int out_index = yIndex + height * xIndex;
    shMat[threadIdx.x][threadIdx.y] = idata[in_index];
    __syncthreads();
    odata[out_index] = shMat[threadIdx.x][threadIdx.y];
}

__global__ void transposeSharedMemNoBanking(float* odata, float* idata, int width, int height) {
    __shared__ float shMat[TILE_DIM][TILE_DIM + 1];
    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    int in_index = xIndex + width * yIndex;
    int out_index = yIndex + height * xIndex;
    shMat[threadIdx.x][threadIdx.y] = idata[in_index];
    __syncthreads();
    odata[out_index] = shMat[threadIdx.x][threadIdx.y];
}




int main()
{
    
    srand((int)time(0));

    int width = TILE_DIM * 16;
    int heigh = TILE_DIM * 8;

    MatrixXf midata = MatrixXf::Zero(heigh, width); // row = heigh  col = width
    auto modata = midata.transpose();


    float* indata = static_cast<float*>(malloc(width * heigh * sizeof(float)));
    for (int i = 0; i < width * heigh; ++i) {
        indata[i] = (float)rand() / (float)INT32_MAX;
        midata(i / width, i % width) = indata[i];
    }
    float* odata = static_cast<float*>(malloc(width * heigh * sizeof(float)));


    hipError_t cudaStatus = transposeWithCuda(odata, indata, width, heigh);
    if (getError(cudaStatus, "transposeWithCuda failed")) return 1;
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (getError(cudaStatus, "hipDeviceReset failed")) return 1;

    // Test
    for (int i = 0; i < width * heigh; ++i) {
        if (modata(i / heigh, i % heigh) != odata[i]) {
            cout << "transposeWithCuda is wrong" << endl;
            goto transposeSharedMemWithCudaTest;
        }
    }
    cout << "transposeWithCuda succeed" << endl;

transposeSharedMemWithCudaTest:

    cudaStatus = transposeSharedMemWithCuda(odata, indata, width, heigh);
    if (getError(cudaStatus, "transposeSharedMemWithCuda failed")) return 1;
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (getError(cudaStatus, "hipDeviceReset failed")) return 1;

    for (int i = 0; i < width * heigh; ++i) {
        if (modata(i / heigh, i % heigh) != odata[i]) {
            cout << "transposeSharedMemWithCuda is wrong" << endl;
            goto transposeSharedMemNoBankingWithCuda;
        }
    }
    cout << "transposeSharedMemWithCuda succeed" << endl;



transposeSharedMemNoBankingWithCuda:
    cudaStatus = transposeSharedMemNoBankingWithCuda(odata, indata, width, heigh);
    if (getError(cudaStatus, "transposeSharedMemNoBankingWithCuda failed")) return 1;
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (getError(cudaStatus, "hipDeviceReset failed")) return 1;

    for (int i = 0; i < width * heigh; ++i) {
        if (modata(i / heigh, i % heigh) != odata[i]) {
            cout << "transposeSharedMemNoBankingWithCuda is wrong" << endl;
            goto End;
        }
    }
    cout << "transposeSharedMemNoBankingWithCuda succeed" << endl;


End:
    return 0;
}

static bool getError(hipError_t cudaStatus, char* message) {
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, message);
        return true;
    }
    return false;
}



hipError_t transposeSharedMemNoBankingWithCuda(float* odata, float* idata, int width, int height) {
    float* dev_odata;
    float* dev_indata;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    

    cudaStatus = hipMalloc((void**)&dev_indata, width * height * sizeof(float));
    

    cudaStatus = hipMalloc((void**)&dev_odata, width * height * sizeof(float));
    

    cudaStatus = hipMemcpy(dev_indata, idata, height * width * sizeof(float), hipMemcpyHostToDevice);
    

    dim3 dimGrid(width / TILE_DIM, height / TILE_DIM);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    transposeSharedMemNoBanking << < dimGrid, dimBlock >> > (dev_odata, dev_indata, width, height);
    cudaStatus = hipGetLastError();


    // wait all done
    cudaStatus = hipDeviceSynchronize();

    // copy back
    cudaStatus = hipMemcpy(odata, dev_odata, width * height * sizeof(float), hipMemcpyDeviceToHost);



    hipFree(dev_indata);
    hipFree(dev_odata);

    return cudaStatus;
}


hipError_t transposeSharedMemWithCuda(float* odata, float* idata, int width, int height) {
    float* dev_odata;
    float* dev_indata;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    

    cudaStatus = hipMalloc((void**)&dev_indata, width * height * sizeof(float));
    

    cudaStatus = hipMalloc((void**)&dev_odata, width * height * sizeof(float));
   

    cudaStatus = hipMemcpy(dev_indata, idata, height * width * sizeof(float), hipMemcpyHostToDevice);


    dim3 dimGrid(width / TILE_DIM, height / TILE_DIM);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    transposeSharedMem << < dimGrid, dimBlock >> > (dev_odata, dev_indata, width, height);
    cudaStatus = hipGetLastError();


    // wait all done
    cudaStatus = hipDeviceSynchronize();


    // copy back
    cudaStatus = hipMemcpy(odata, dev_odata, width * height * sizeof(float), hipMemcpyDeviceToHost);



Error:
    hipFree(dev_indata);
    hipFree(dev_odata);

    return cudaStatus;
}


hipError_t transposeWithCuda(float* odata, float* idata, int width, int height) {
    float* dev_odata;
    float* dev_indata;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);


    cudaStatus = hipMalloc((void**)&dev_indata, width * height * sizeof(float));

    cudaStatus = hipMalloc((void**)&dev_odata, width * height * sizeof(float));


    cudaStatus = hipMemcpy(dev_indata, idata, height * width * sizeof(float), hipMemcpyHostToDevice);


    dim3 dimGrid(width / TILE_DIM, height / TILE_DIM);
    dim3 dimBlock(TILE_DIM, TILE_DIM);

    transposeNaive << < dimGrid, dimBlock >> > (dev_odata, dev_indata, width, height);
    cudaStatus = hipGetLastError();


    // wait all done
    cudaStatus = hipDeviceSynchronize();


    // copy back
    cudaStatus = hipMemcpy(odata, dev_odata, width * height * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_indata);
    hipFree(dev_odata);

    return cudaStatus;
}